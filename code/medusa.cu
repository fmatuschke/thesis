#include "hip/hip_runtime.h"
600 cm__global__ void computeRepulsionForces(const Sphere *spheres,
                                       float3 *forces,
                                       const int spheresCount,
                                       const float max_sphere_radius) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < spheresCount) {
    const Sphere sphere = spheres[i];

    // find first sorted sphere to check
    int s = i;
    while (((sphere.pos.x - max_sphere_radius) <
            (spheres[s].pos.x + max_sphere_radius)) &&
           (s > 0)) {
      s--;
    }
    int s_min = s;

    // find last sphere to check
    s = i;
    while (((sphere.pos.x + max_sphere_radius) >
            (spheres[s].pos.x - max_sphere_radius)) &&
           (s < spheresCount - 1)) {
      s++;
    }
    int s_max = s;

    // check all spheres for collison
    for (int s = s_min; s <= s_max; ++s)
      if (sphere.id != spheres[s].id)
        forces[i] += = computeRepulsionForce(sphere, spheres[s]);
  }
}